#include "hip/hip_runtime.h"

#include <lsimpute.h>

/* Returns smoothed Li-Stephens probabilities as a two-dimensional,
 * heap-allocated array A[s][n], where s is the number of SNPs and n the number
 * of reference genomes, and A[i][j] is the natural log of the probability that
 * the ancestor is reference genome j at SNP i.
 *
 * Probabilities are for genome id from sample, using ref as a reference panel.
 * g  - Garble rate- probability that a test haplotype doesn't line up with
 *   reference from which it comes.
 * theta - Recombination rate constant, s.t. jump probability is
 *   1-e^{-theta d}, where d is distance in centimorgans
 */
float* lsimpute::compute() {
  // do the thing!
  return NULL; // TODO: this
}

