#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "lsimpute.h"

#define EMISS(o1, o2, g) log(o1 == o2 ? (1 - g) : g)

#define BLOCKMAX 1024 // TODO: set to 1024 if compute capability >= 2.0

/* Calculates log(exp(x) + exp(y))
 */
__device__ float d_logadd(float x, float y) {
  if (y > x) { return d_logadd(y,x); }
  return x + logf(1+expf(y-x));
}

/* Calculates log(1.0f - exp(x))
 */
__device__ float d_logsub1(float x) {
  return logf(1.0f - expf(x));
}

__device__ bool isPow2(int n) {
  return (n != 0) && (n & (n-1)) == 0;
}

/* Sums the n floating point values in array A (in no particular order)
 * that are in natural log space
 * Basically, find log(sum(p1 ... pn)) given log(p1) .. log(pn)
 */
__device__ float row_logsum(float* A, int n, float* scratch) {
  int tid = threadIdx.x;
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  int elts = n / BLOCKMAX + (tid < n % BLOCKMAX);

  // With sample sizes being single-digit multiples of BLOCKMAX (at least at
  // the moment), we can get away with doing this linearly. As sample sizes
  // get larger, we'll need to start looking into recursive kernel invocation.
  scratch[tid] = A[i];
  for (int j = 1 ; j < elts ; j += 1) {
    scratch[tid] = d_logadd(scratch[tid], A[i+j]);
  }
  __syncthreads();

  for (int s = 1 ; s < blockDim.x ; s <<= 1) {
    int index = 2*s*tid;
    if (index < blockDim.x) {
      scratch[tid] = d_logadd(scratch[index], scratch[index+s]);
    }
  }
  __syncthreads();

  return scratch[0];
}

/* Normalizes the n floating point values in the array starting at A
 * (sets A[i] = log(exp(A[i]) / exp(reduce_logsum(A,n))))
 */
__device__ void d_logrownorm(float* A, int n, float* scratch) {
  float x = row_logsum(A, n, scratch);
  for (int i = 0; i < n; i++) A[i] -= x;
  return;
}

__device__ void fwKernel(uint8_t* refs, uint8_t* sample, float* dists,
    float* fw, float g, float theta, int nsnp, int nsample, float* scratch) {
  // Initialize first row
  float c = 1.0f / ((float)nsample);
  for (int i = threadIdx.x; i < nsample; i += blockDim.x) fw[i] = c;
  __syncthreads();

  // For each SNP (going forward)
  for (int k = 1; k < nsnp; k++) {
    int K = k * nsample;
    // Precompute jump probability
    int J = row_logsum(&(fw[K]), nsample, scratch);
    J = J + d_logsub1(-1.0f * theta * dists[k]);
    float nJ = d_logsub1(J);

    // Calculate values
    for (int i = threadIdx.x; i < nsample; i += blockDim.x) {
      float alpha = d_logadd(fw[K - nsample + i] + nJ, J + c);
      fw[K + i] = alpha + EMISS(sample[k], refs[K+i],g);
    }
  }
  return;
}

__device__ void bwKernel(uint8_t* refs, uint8_t* sample, float* dists,
    float* bw, float g, float theta, int nsnp, int nsample, float* scratch) {
  // Initialize last row
  float c = 1.0f / ((float)nsample);
  for (int i = threadIdx.x; i < nsample; i += blockDim.x) {
    bw[(nsample * (nsnp - 1)) + i] =
      c + EMISS(refs[nsample * (nsnp - 1) + i], sample[nsnp - 1], g);
  }

  // For each SNP (going backward)
  for (int k = nsnp - 2; k >= 0; k--) {
    int K = k * nsample;
    // Precompute jump probability
    float J = row_logsum(&(bw[K+nsample]),nsample, scratch);
    J = J + d_logsub1(-1.0f * theta * dists[k]);
    float nJ = d_logsub1(J);

    // Calculate values
    for (int i = threadIdx.x; i < nsample; i += blockDim.x) {
      float alpha = d_logadd(J + c, nJ + bw[K + i + nsample]);
      bw[K + i] = alpha + EMISS(sample[k], refs[K+i], g);
    }
  }
  return;
}

/* Returns its smoothed answer in fw
 */
__device__ void smoothKernel(
    float* fw, float* bw, int nsnp, int nsample, float* scratch
) {
  for (int i = 0; i < nsnp; i++) {
    int I = i * nsample;
    for (int j = 0; j < nsample; j++) {
      fw[I + j] = fw[I + j] + bw[I + j];
    }
    d_logrownorm(&(fw[I]), nsample, scratch);
  }
  return;
}

__global__ void computeKernel(uint8_t* refs, uint8_t* sample, float* dists,
    float* fw, float* bw, float g, float theta, int nsnp, int nsample) {
  extern __shared__ float scratch[];
  // Forward step
  fwKernel(refs, sample, dists, fw, g, theta, nsnp, nsample, scratch);

  // Backward step
  bwKernel(refs, sample, dists, bw, g, theta, nsnp, nsample, scratch);

  // Smoothing step
  smoothKernel(fw, bw, nsnp, nsample, scratch);
  return;
}

float* lsimputer::compute(uint8_t* snps) {
  // Allocate space for refs, sample, distances, and return values
  uint8_t* d_refs;
  uint8_t* d_sample;
  float* d_dists;
  float* d_fw;
  float* d_bw;
  hipMalloc((void **)&d_refs, sizeof(uint8_t) * nsnp * nsample);
  hipMalloc((void **)&d_sample, sizeof(uint8_t) * nsnp);
  hipMalloc((void **)&d_dists, sizeof(float) * nsnp);
  hipMalloc((void **)&d_fw, sizeof(float) * nsnp * nsample);
  hipMalloc((void **)&d_bw, sizeof(float) * nsnp * nsample);

  // Transfer over data
  hipMemcpy(d_refs, ref, sizeof(uint8_t) * nsnp * nsample,
      hipMemcpyHostToDevice);
  hipMemcpy(d_sample, snps, sizeof(uint8_t) * nsnp,
      hipMemcpyHostToDevice);
  hipMemcpy(d_dists, dists, sizeof(float) * nsnp,
      hipMemcpyHostToDevice);

  // Run the kernel
  computeKernel<<<1, BLOCKMAX, nsample*sizeof(float)>>>
    (d_refs, d_sample, d_dists, d_fw,
      d_bw, g, theta, nsnp, nsample);
  hipDeviceSynchronize();

  // Transfer data off the device
  float* res = (float*)malloc(sizeof(float) * nsnp * nsample);
  hipMemcpy(res, d_fw, sizeof(uint8_t) * nsnp * nsample,
      hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_refs);
  hipFree(d_sample);
  hipFree(d_dists);
  hipFree(d_fw);
  hipFree(d_bw);

  return res;
}

