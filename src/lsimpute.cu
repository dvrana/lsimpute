#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include "lsimpute.h"

#define EMISS(o1, o2, g) (o1 == o2 ? (1 - g) : g)

#define BLOCKMAX 512 // TODO: set to 1024 if compute capability >= 2.0
#define WARP_SIZE 32

__device__ bool isPow2(int n) {
  return (n != 0) && (n & (n-1)) == 0;
}

__device__ __host__
int npow2(int n) {
  n -= 1;
  n |= n >> 1;
  n |= n >> 2;
  n |= n >> 4;
  n |= n >> 8;
  n |= n >> 16;
  return n+1;
}

/* Sums the n floating point values in array A (in no particular order)
 * that are in natural log space
 * Basically, find log(sum(p1 ... pn)) given log(p1) .. log(pn)
 */
__device__ float row_sum(float* A, int n, float* scratch) {
  n = npow2(n);
  // Possible performance speedup: half our threads are idle on the first loop!
  int tid = threadIdx.x;
  int nthread = blockDim.x;
  int fakeelts = n / nthread;

  int i = blockIdx.x*blockDim.x + (fakeelts * tid) + min(n % nthread, tid);
  int elts = fakeelts + (tid < n % nthread);

  // With sample sizes being single-digit multiples of BLOCKMAX (at least at
  // the moment), we can get away with doing this linearly. As sample sizes
  // get larger, we'll need to start looking into recursive kernel invocation.
  scratch[tid] = A[i];
  for (int j = 1 ; j < elts && i+j < n ; j += 1) {
    scratch[tid] = scratch[tid] + A[i+j];
  }
  __syncthreads();

  int big = n > WARP_SIZE;

  for (int s = nthread/2 ; s > WARP_SIZE*big ; s >>= 1) {
    if (tid < s && tid+s < n) {
      scratch[tid] = scratch[tid] + scratch[tid+s];
    }
    __syncthreads();
  }

  if (tid < WARP_SIZE && big) {
    #pragma unroll
    for (int j = WARP_SIZE ; j > 0 ; j >>= 1) {
      if (tid+j < n)
        scratch[tid] = scratch[tid] + scratch[tid+j];
    }
  }

  __syncthreads();
  return scratch[0];
}

/* Normalizes the n floating point values in the array starting at A
 * (sets A[i] = log(exp(A[i]) / exp(reduce_logsum(A,n))))
 */
__device__ void d_rownorm(float* A, int n, float* scratch) {
  float x = row_sum(A, n, scratch);
  for (int i = 0; i < n; i++) A[i] /= x;
  return;
}

__device__ void fwKernel(uint8_t* refs, uint8_t* sample, float* dists,
    float* fw, float g, float theta, int nsnp, int nsample, float* scratch) {
  // Initialize first row
  float c = 1.0f / ((float)nsample);
  for (int i = threadIdx.x; i < nsample; i += blockDim.x)
    fw[i] = EMISS(sample[0], refs[i],g);
  __syncthreads();

  // For each SNP (going forward)
  for (int k = 1; k < nsnp; k++) {
    int K = k * nsample;
    // Precompute jump probability
    float x = row_sum(&(fw[K-nsample]), nsample, scratch);
    float nJ = expf(-1.0f * theta * dists[k-1]);
    float J = 1.0 - nJ;

    // Calculate values
    for (int i = threadIdx.x; i < nsample; i += blockDim.x) {
      fw[K - nsample + i] /= x;
      float alpha = fw[K - nsample + i] * nJ + J * c;
      fw[K + i] = alpha * EMISS(sample[k], refs[K+i],g);
    }
    __syncthreads();
  }
  return;
}

__device__ void bwKernel(uint8_t* refs, uint8_t* sample, float* dists,
    float* bw, float g, float theta, int nsnp, int nsample, float* scratch) {
  // Initialize last row
  float c = 1.0f / ((float)nsample);
  for (int i = threadIdx.x; i < nsample; i += blockDim.x) {
    bw[(nsample * (nsnp - 1)) + i] =
      EMISS(refs[nsample * (nsnp - 1) + i], sample[nsnp - 1], g);
  }

  // For each SNP (going backward)
  for (int k = nsnp - 2; k >= 0; k--) {
    int K = k * nsample;
    // Precompute jump probability
    float x = row_sum(&(bw[K+nsample]), nsample, scratch);
    float nJ = expf(-1.0f * theta * dists[k]);
    float J = 1.0f - nJ;

    // Calculate values
    for (int i = threadIdx.x; i < nsample; i += blockDim.x) {
      bw[K + nsample + i] /= x;
      float alpha = (J * c) + (nJ * bw[K + i + nsample]);
      bw[K + i] = alpha * EMISS(sample[k], refs[K+i], g);
    }
    __syncthreads();
  }
  return;
}

/* Returns its smoothed answer in fw
 */
__device__ void smoothKernel(
    float* fw, float* bw, int nsnp, int nsample, float* scratch) {
  for (int i = 0; i < nsnp-1; i++) {
    int I = i * nsample;
    for (int j = threadIdx.x; j < nsample; j += blockDim.x) {
      fw[I + j] = fw[I + j] * bw[I + nsample + j];
    }
    __syncthreads();
    d_rownorm(fw + I, nsample, scratch);
  }
  d_rownorm(fw + (nsample * (nsnp-1)), nsample, scratch);
  return;
}

__global__ void computeKernel(uint8_t* refs, uint8_t* sample, float* dists,
    float* fw, float* bw, float g, float theta, int nsnp, int nsample,
    int nscratch) {
  extern __shared__ float scratch[];

  __syncthreads();

  // Forward step
  fwKernel(refs, sample, dists, fw, g, theta, nsnp, nsample, scratch);

  // Backward step
  bwKernel(refs, sample, dists, bw, g, theta, nsnp, nsample, scratch);

  // Smoothing step
  smoothKernel(fw, bw, nsnp, nsample, scratch);
  return;
}

float* lsimputer::compute(uint8_t* snps) {
  // Allocate space for refs, sample, distances, and return values
  uint8_t* d_refs;
  uint8_t* d_sample;
  float* d_dists;
  float* d_fw;
  float* d_bw;
  hipMalloc((void **)&d_refs, sizeof(uint8_t) * nsnp * nsample);
  hipMalloc((void **)&d_sample, sizeof(uint8_t) * nsnp);
  hipMalloc((void **)&d_dists, sizeof(float) * nsnp);
  hipMalloc((void **)&d_fw, sizeof(float) * nsnp * nsample);
  hipMalloc((void **)&d_bw, sizeof(float) * nsnp * nsample);

  // Transfer over data
  hipMemcpy(d_refs, ref, sizeof(uint8_t) * nsnp * nsample,
      hipMemcpyHostToDevice);
  hipMemcpy(d_sample, snps, sizeof(uint8_t) * nsnp,
      hipMemcpyHostToDevice);
  hipMemcpy(d_dists, dists, sizeof(float) * nsnp,
      hipMemcpyHostToDevice);

  int nthread = npow2(min(BLOCKMAX, max(nsample, 32)));
  int nscratch = max(nthread, npow2(nsample));

  // Run the kernel
  computeKernel<<<1, nthread, nscratch*sizeof(float)>>>
    (d_refs, d_sample, d_dists, d_fw,
      d_bw, g, theta, nsnp, nsample, nscratch);
  hipDeviceSynchronize();

  // Transfer data off the device
  float* res = (float*)malloc(sizeof(float) * nsnp * nsample);
  hipMemcpy(res, d_fw, sizeof(float) * nsnp * nsample,
      hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_refs);
  hipFree(d_sample);
  hipFree(d_dists);
  hipFree(d_fw);
  hipFree(d_bw);

  return res;
}

