#include "hip/hip_runtime.h"

#include "lsimpute.h"

#define BLOCKMAX 512 // TODO: set to 1024 if compute capability >= 2.0

__global__ void computeKernel(uint8_t* refs, uint8_t* sample, float* dists,
    float* fw, float* bw, float g, float theta) {
  return; // TODO: this
}

float* lsimputer::compute(uint8_t* snps) {
  // Allocate space for refs, sample, distances, and return values
  uint8_t* d_refs;
  uint8_t* d_sample;
  float* d_dists;
  float* d_fw;
  float* d_bw;
  hipMalloc((void **)&d_refs, sizeof(uint8_t) * nsnp * nsample);
  hipMalloc((void **)&d_sample, sizeof(uint8_t) * nsnp);
  hipMalloc((void **)&d_dists, sizeof(float) * nsnp);
  hipMalloc((void **)&d_fw, sizeof(float) * nsnp * nsample);
  hipMalloc((void **)&d_bw, sizeof(float) * nsnp * nsample);

  // Transfer over data
  hipMemcpy(d_refs, ref, sizeof(uint8_t) * nsnp * nsample,
      hipMemcpyHostToDevice);
  hipMemcpy(d_sample, snps, sizeof(uint8_t) * nsnp,
      hipMemcpyHostToDevice);
  hipMemcpy(d_dists, dists, sizeof(float) * nsnp,
      hipMemcpyHostToDevice);

  // Run the kernel
  computeKernel<<<1, BLOCKMAX>>>(d_refs, d_sample, d_dists, d_fw, d_bw, g, theta);
  hipDeviceSynchronize();

  // Transfer data off the device
  float* res = (float*)malloc(sizeof(float) * nsnp * nsample);
  hipMemcpy(res, d_fw, sizeof(uint8_t) * nsnp * nsample,
      hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_refs);
  hipFree(d_sample);
  hipFree(d_dists);
  hipFree(d_fw);
  hipFree(d_bw);

  return res;
}

