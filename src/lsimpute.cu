#include "hip/hip_runtime.h"

#include "lsimpute.h"

#define EMISS(o1, o2, g) log(o1 == o2 ? (1 - g) : g)

#define BLOCKMAX 512 // TODO: set to 1024 if compute capability >= 2.0

/* Sums the n floating point values in array A (in no particular order)
 * that are in natural log space
 * Basically, find log(sum(p1 ... pn)) given log(p1) .. log(pn)
 */
__device__ float reduce_logsum(float* A, int n) {
  return logf(420.0f); // TODO: This (for Cam)
}

/* Calculates log(exp(x) + exp(y))
 */
__device__ float d_logadd(float x, float y) {
  return 0.0f; // TODO: this
}

/* Calculates log(1.0f - exp(x))
 */
__device__ float d_logsub1(float x) {
  return 0.0f; // TODO: this
}

__device__ void fwKernel(uint8_t* refs, uint8_t* sample, float* dists,
    float* fw, float g, float theta, int nsnp, int nsample) {
  // Initialize first row
  float c = 1.0f / ((float)nsample);
  for (int i = threadIdx.x; i < nsample; i += blockDim.x) fw[i] = c;

  // For each SNP (going forward)
  for (int k = 1; k < nsnp; k++) {
    int K = k * nsample;
    // Precompute jump probability
    float J = reduce_logsum(&(fw[K]), nsample);
    J = J + d_logsub1(-1.0f * theta * dists[k]);
    float nJ = d_logsub1(J);

    // Calculate values
    for (int i = threadIdx.x; i < nsample; i += blockDim.x) {
      float alpha = d_logadd(fw[K - nsample + i] + nJ, J + c);
      fw[K + i] = alpha + EMISS(sample[k], refs[K+i],g);
    }
  }
  return;
}

__device__ void bwKernel(uint8_t* refs, uint8_t* sample, float* dists,
    float* bw, float g, float theta, int nsnp, int nsample) {
  // Initialize last row
  float c = 1.0f / ((float)nsample);
  for (int i = threadIdx.x; i < nsample; i += blockDim.x) {
    bw[(nsample * (nsnp - 1)) + i] =
      c + EMISS(refs[nsample * (nsnp - 1) + i], sample[nsnp - 1], g);
  }

  // For each SNP (going backward)
  for (int k = nsnp - 2; k >= 0; k--) {
    int K = k * nsample;
    // Precompute jump probability
    float J = reduce_logsum(&(bw[K+nsample]),nsample);
    J = J + d_logsub1(-1.0f * theta * dists[k]);
    float nJ = d_logsub1(J);

    // Calculate values
    for (int i = threadIdx.x; i < nsample; i += blockDim.x) {
      float alpha = d_logadd(J + c, nJ + bw[K + i + nsample]);
      bw[K + i] = alpha + EMISS(sample[k], refs[K+i], g);
    }
  }
  return;
}


__device__ void smoothKernel(float* fw, float* bw, int nsnp, int nsample) {
  return; // TODO: this
}

__global__ void computeKernel(uint8_t* refs, uint8_t* sample, float* dists,
    float* fw, float* bw, float g, float theta, int nsnp, int nsample) {
  // Forward step
  fwKernel(refs, sample, dists, fw, g, theta, nsnp, nsample);

  // Backward step
  bwKernel(refs, sample, dists, bw, g, theta, nsnp, nsample);

  // Smoothing step
  smoothKernel(fw, bw, nsnp, nsample);
  return;
}

float* lsimputer::compute(uint8_t* snps) {
  // Allocate space for refs, sample, distances, and return values
  uint8_t* d_refs;
  uint8_t* d_sample;
  float* d_dists;
  float* d_fw;
  float* d_bw;
  hipMalloc((void **)&d_refs, sizeof(uint8_t) * nsnp * nsample);
  hipMalloc((void **)&d_sample, sizeof(uint8_t) * nsnp);
  hipMalloc((void **)&d_dists, sizeof(float) * nsnp);
  hipMalloc((void **)&d_fw, sizeof(float) * nsnp * nsample);
  hipMalloc((void **)&d_bw, sizeof(float) * nsnp * nsample);

  // Transfer over data
  hipMemcpy(d_refs, ref, sizeof(uint8_t) * nsnp * nsample,
      hipMemcpyHostToDevice);
  hipMemcpy(d_sample, snps, sizeof(uint8_t) * nsnp,
      hipMemcpyHostToDevice);
  hipMemcpy(d_dists, dists, sizeof(float) * nsnp,
      hipMemcpyHostToDevice);

  // Run the kernel
  computeKernel<<<1, BLOCKMAX>>>(d_refs, d_sample, d_dists, d_fw, d_bw, g,
      theta, nsnp, nsample);
  hipDeviceSynchronize();

  // Transfer data off the device
  float* res = (float*)malloc(sizeof(float) * nsnp * nsample);
  hipMemcpy(res, d_fw, sizeof(uint8_t) * nsnp * nsample,
      hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_refs);
  hipFree(d_sample);
  hipFree(d_dists);
  hipFree(d_fw);
  hipFree(d_bw);

  return res;
}

